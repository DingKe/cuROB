#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hiprand.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define EXCLUDE
#include "cuROB.h"
#undef EXCLUDE

#pragma comment(lib,"cublas.lib")

#define CHECK_CUDA_ERROR()                                                                                \
{                                                                                                         \
	hipError_t err = hipGetLastError();                                                                   \
	if (err != hipSuccess)                                                                                 \
  {                                                                                                       \
  printf("%s.%s.%d: Error %d - %s.\n", __FILE__, __FUNCTION__, __LINE__, err, hipGetErrorString(err)); \
  \
  }                                                                                                       \
}                                                                                                         \

static int a = 1111;
int irand()
{
	a =  ( a * 1103515245 + 12345 ) & 0x7fffffff;
	return a;
}
double drand()
{
	return (double)(irand()+1.0)/(0x7fffffff+1.0);
}


/*
** Always be 32 for the current CUDA-enabled GPUs
*/
#define WARP_SIZE 32
/*
** BLOCK_SIZE_EVALUATION should be equal to WARP_SIZE in the implementation
*/
#define BLOCK_SIZE_EVALUATION WARP_SIZE

#define MAX_COMPOSITION_NUM (5)
#define SUBCOMPONENT_NUM (3)


/*
** Miscellaneous
*/
#define SWAP(x,y) { int tem = (y); (y) = (x); (x) = tem;}
#define DIVUP(x,y) ( ((x)+(y)-1)/(y) )

#define PI (3.1415926535897932384626433832795029)
#define E (2.7182818284590452353602874713526625)

#define BIAS (100)
#define OPTIMA_RANGE (40)

/*
** Data for composition functions
*/
static int COMPOSITION_NUMS[] = {5,3,3,5,5,5,3,3};

static double bias[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{0,100,200,300,400},{0,100,200},{0,100,200},{0,100,200,300,400},{0,100,200,300,400},{0,100,200,300,400},{0,100,200},{0,100,200}
};
static double lamda[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{1,1e-6,1e-26,1e-6,1e-6},{1,1,1},{0.25,1,1e-7},{0.25,1,1e-7,2.5,10},{10,10,2.5,2.5,1e-6},{2.5,10,2.5,5e-4,1e-6},{1,1,1},{1,1,1}
};
static double w[MAX_COMPOSITION_NUM*MAX_CONCURRENCY];
static double sigma[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{10,20,30,40,50},{20,20,20},{10,30,50},{10,10,10,10,10},{10,10,10,20,20},{10,20,30,40,50},{10,30,50},{10,30,50}
};

static float biasf[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{0,100,200,300,400},{0,100,200},{0,100,200},{0,100,200,300,400},{0,100,200,300,400},{0,100,200,300,400},{0,100,200},{0,100,200}
};
static float lamdaf[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{1,1e-6f,1e-26f,1e-6f,1e-6f},{1,1,1},{0.25f,1,1e-7f},{0.25f,1,1e-7f,2.5f,10},{10,10,2.5,2.5,1e-6f},{2.5f,10,2.5f,5e-4f,1e-6f},{1,1,1},{1,1,1}
};
static float wf[MAX_COMPOSITION_NUM*MAX_CONCURRENCY];
static float sigmaf[COMPOSITION_FUNC_NUM][MAX_COMPOSITION_NUM] = {
	{10,20,30,40,50},{20,20,20},{10,30,50},{10,10,10,10,10},{10,10,10,20,20},{10,20,30,40,50},{10,30,50},{10,30,50}
};

/*
** Pre-allocated Memory Space
*/
static double *o, *m, *optima, *bufferA, *bufferB, *bufferC, *omega, *devx, *devy;
static float *of, *mf, *optimaf, *bufferAf, *bufferBf, *bufferCf, *omegaf, *devxf, *devyf;


/*
** structure for CUBLAS
*/
static hipblasHandle_t handle;

static bool initialized = false;

#ifdef __cplusplus
extern "C" {
#endif

	void curob_initialize()
	{	
		if( initialized )
		{
			return;
		}

		hipblasCreate(&handle);

		hipMalloc(&o, sizeof(double)*BASIC_FUNC_NUM*DIM);
		hipMalloc(&of, sizeof(float)*BASIC_FUNC_NUM*DIM);
		hipMalloc(&m,sizeof(double)*BASIC_FUNC_NUM*DIM*DIM);
		hipMalloc(&mf,sizeof(float)*BASIC_FUNC_NUM*DIM*DIM);

		hipMalloc(&optima,sizeof(double)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM);
		hipMalloc(&optimaf,sizeof(float)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM);

		hipMalloc(&omega,sizeof(double)*MAX_CONCURRENCY*MAX_COMPOSITION_NUM);
		hipMalloc(&omegaf,sizeof(float)*MAX_CONCURRENCY*MAX_COMPOSITION_NUM);

		hipMalloc(&bufferA,sizeof(double)*DIM*MAX_CONCURRENCY);
		hipMalloc(&bufferAf,sizeof(float)*DIM*MAX_CONCURRENCY);
		hipMalloc(&bufferB,sizeof(double)*DIM*MAX_CONCURRENCY);
		hipMalloc(&bufferBf,sizeof(float)*DIM*MAX_CONCURRENCY);
		hipMalloc(&bufferC,sizeof(double)*MAX_CONCURRENCY*MAX_COMPOSITION_NUM);
		hipMalloc(&bufferCf,sizeof(float)*MAX_CONCURRENCY*MAX_COMPOSITION_NUM);

		hipMalloc(&devx,sizeof(double)*DIM*MAX_CONCURRENCY);
		hipMalloc(&devxf,sizeof(float)*DIM*MAX_CONCURRENCY);

		hipMalloc(&devy,sizeof(double)*MAX_CONCURRENCY);
		hipMalloc(&devyf,sizeof(float)*MAX_CONCURRENCY);


		CHECK_CUDA_ERROR();	


		double *h_o = (double*)malloc(sizeof(double)*BASIC_FUNC_NUM*DIM);
		float *h_of = (float*)malloc(sizeof(float)*BASIC_FUNC_NUM*DIM);
		double *h_m = (double*)malloc(sizeof(double)*BASIC_FUNC_NUM*DIM*DIM);
		float *h_mf = (float*)malloc(sizeof(float)*BASIC_FUNC_NUM*DIM*DIM);

		int i, j, k, n;

		/* Generate shifting data randomly */
		for( i = 0; i < BASIC_FUNC_NUM; i++ )
		{
			for( j = 0; j < DIM; j++)
			{			
				h_o[i*DIM+j] = (drand()-0.5)*2*OPTIMA_RANGE;
				h_of[i*DIM+j] = (float)h_o[i*DIM+j];			
			}
		}
		hipMemcpy(o,h_o,sizeof(double)*BASIC_FUNC_NUM*DIM,hipMemcpyHostToDevice);
		CHECK_CUDA_ERROR();
		hipMemcpy(of,h_of,sizeof(float)*BASIC_FUNC_NUM*DIM,hipMemcpyHostToDevice);	
		CHECK_CUDA_ERROR();

		/* Generate rotation data randomly */
		double x1, x2;
		double prod, tem[DIM], * mptr;

		for ( n = 0; n < BASIC_FUNC_NUM; n++ )
		{
			int x[DIM];
			int y[DIM];
			j = 0;

			if( n+1 == HYBRID1 || n+1 == HYBRID2 )
			{
				for( i = 0; i < DIM; i++)
				{
					if ( i < ceil(0.3*DIM) )	
						x[i] = y[i] = 0;
					else if ( i < ceil(0.3*DIM)*2 )
						x[i] = y[i] = 1;
					else
						x[i] = y[i] = 2;
				}
			}
			else if ( n+1 == HYBRID3 || n+1 == HYBRID4 )
			{
				for ( i = 0; i < DIM; i++ )
				{
					if ( i < ceil(0.2*DIM) )	
						x[i] = y[i] = 0;
					else if ( i < ceil(0.2*DIM)*2 )
						x[i] = y[i] = 1;
					else if ( i < ceil(0.3*DIM)+ceil(0.2*DIM)*2 )
						x[i] = y[i] = 2;
					else
						x[i] = y[i] = 3;
				}
			}
			else if( n+1 == HYBRID5 || n+1 == HYBRID6 )
			{
				for ( i = 0; i < DIM; i++)
				{
					if ( i < ceil(0.1*DIM) )	
						x[i] = y[i] = 0;
					else if ( i < ceil(0.2*DIM)+ceil(0.1*DIM) )
						x[i] = y[i] = 1;
					else if ( i < ceil(0.2*DIM)*2+ceil(0.1*DIM) )
						x[i] = y[i] = 2;
					else if ( i < ceil(0.2*DIM)*3+ceil(0.1*DIM) )
						x[i] = y[i] = 3;
					else
						x[i] = y[i] = 4;
				}	
			} else // non-hybrid functions
			{
				for ( i = 0; i < SUBCOMPONENT_NUM; i++ )
				{
					for ( j = i*DIM/SUBCOMPONENT_NUM; j < (i+1)*DIM/SUBCOMPONENT_NUM; j++ )
					{
						x[j] = i;
						y[j] = i;
					}			
				}
			}

			/* Permutation */
			for ( i = 0; i < DIM; i++ )
			{
				for ( j = 0; j < DIM; j++ )
				{
					if ( drand() > 0.5 )
					{
						SWAP(x[i],x[j]);
					}
					if ( drand() > 0.5 && (n < HYBRID1 || n > HYBRID6) )
					{
						SWAP(y[i],y[j]);
					}
				}
			}

			mptr = &h_m[n*DIM*DIM];		
			for ( i = 0; i < DIM; i++ )
			{
				for ( j = 0; j < DIM; j++ )
				{
					if ( x[i] == y[j] )
					{
						/*  Box-Muller for normal distribution */				
						x1 = drand();				
						x2 = drand();;				
						mptr[i*DIM+j] = sqrt(-2.0*log(x1))*cos(2*PI*x2);
					}
					else
					{
						mptr[i*DIM+j] = 0;
					}
				}
			}		 
		}

		for ( n = 0; n < BASIC_FUNC_NUM; n++ )
		{
			mptr = &h_m[n*DIM*DIM];
			/* Gram-Schmidt Orthonormalization */
			for ( i = 0; i < DIM; i++)
			{
				/* Orthogonalize */
				for ( k = 0; k < DIM; k++ )
				{
					tem[k] = mptr[i*DIM+k];
				}
				for ( j = 0; j < i; j++ )
				{		
					/* Inner product */
					prod = 0;
					for ( k = 0; k < DIM; k++ )
					{
						prod += mptr[i*DIM+k]*mptr[j*DIM+k];
					}

					for ( k = 0; k < DIM; k++ )
					{
						tem[k] -= prod*mptr[j*DIM+k];
					}
				}

				/* Normalize */
				prod = 0;
				for( k = 0; k < DIM; k++)
				{
					prod += tem[k]*tem[k];
				}	
				prod = sqrt(prod);
				for( k = 0; k < DIM; k++)
				{
					mptr[i*DIM+k] = tem[k]/prod;
				}
			}
		}

		for ( n = 0; n < BASIC_FUNC_NUM; n++ )
		{		
			for( i = 0; i < DIM; i++)
			{
				for( j = 0; j < DIM; j++)
				{
					h_mf[n*DIM*DIM+i*DIM+j] = (float)h_m[n*DIM*DIM+i*DIM+j];
				}
			}
		}	

		hipMemcpy(m,h_m,sizeof(double)*BASIC_FUNC_NUM*DIM*DIM,hipMemcpyHostToDevice);
		CHECK_CUDA_ERROR();
		hipMemcpy(mf,h_mf,sizeof(float)*BASIC_FUNC_NUM*DIM*DIM,hipMemcpyHostToDevice);
		CHECK_CUDA_ERROR();

		double *h_optima = (double*)malloc(sizeof(double)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM);
		float *h_optimaf = (float*)malloc(sizeof(float)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM);

		for( n = 0; n < COMPOSITION_FUNC_NUM; n++ )
		{		
			double * A = h_optima+n*MAX_COMPOSITION_NUM*DIM;
			float * Af = h_optimaf+n*MAX_COMPOSITION_NUM*DIM;
			for( i = 0; i < MAX_COMPOSITION_NUM; i++ )
			{
				for( j = 0; j < DIM; j++)
				{
					A[i*DIM+j] = (drand()-0.5)*2*OPTIMA_RANGE;
				}
			}
			/* set 3rd to 0s */
			for( j = 0; j < DIM; j++)
			{
				A[2*DIM+j] = 0;
			}

			for( i = 0; i < MAX_COMPOSITION_NUM; i++ )
			{
				for( j = 0; j < DIM; j++)
				{
					Af[i*DIM+j] = (float)A[i*DIM+j];
				}
			}
		}

		hipMemcpy( optima, h_optima, sizeof(double)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM, hipMemcpyHostToDevice );
		CHECK_CUDA_ERROR();
		hipMemcpy( optimaf, h_optimaf, sizeof(float)*COMPOSITION_FUNC_NUM*MAX_COMPOSITION_NUM*DIM, hipMemcpyHostToDevice );
		CHECK_CUDA_ERROR();


		//#define VERIFY	
#ifdef VERIFY

		/* Shift Vectors */
		for( n = 0; n < BASIC_FUNC_NUM; n++ )
		{		
			double * A = h_o+n*DIM;		
			printf("No. %d:\n",n+1);		
			for( j = 0; j < DIM; j++)
			{
				printf("%.20f\t", A[j]);
			}
			printf("\n");		
		}


		for( n = 0; n < COMPOSITION_FUNC_NUM; n++ )
		{		
			double * A = h_optima+n*MAX_COMPOSITION_NUM*DIM;		
			printf("No. %d:\n",n+BASIC_FUNC_NUM+1);
			for( i = 0; i < COMPOSITION_NUMS[n]; i++ )
			{
				printf("Composition %d\n",i+1);
				for( j = 0; j < DIM; j++)
				{
					printf("%.20f\t", A[i*DIM+j]);
				}
				printf("\n");
			}
		}


		/* rotation matrice */
		for( n = 0; n < BASIC_FUNC_NUM; n++)
		{
			printf("No. %d:\n",n+1);
			double* A = &h_m[n*DIM*DIM];			
			for( i = 0; i < DIM; i++)
			{
				for( j = 0; j < DIM; j++)
				{
					printf("%.20f\t",A[i*DIM+j]);
				}
				printf("\n");
			}
		}

		/* verify orthogoalism and normalism */
		for( n = 0; n < BASIC_FUNC_NUM; n++)
		{
			printf("No. %d:\n",n+1);
			double* A = &h_m[n*DIM*DIM];			
			for( i = 0; i < DIM; i++)
			{
				for( j = 0; j < DIM; j++)
				{
					/* inner product*/
					double product = 0;
					for( k = 0; k < DIM; k++)
					{
						product += A[i*DIM+k]*A[j*DIM+k];
					}
					printf("%.3f\t",product);
				}
				printf("\n");
			}		
		}	

#endif

		hipDeviceSynchronize();

		free(h_o);
		free(h_of);
		free(h_m);
		free(h_mf);

		free(h_optima);
		free(h_optimaf);


		initialized = true;
		return;
	}

	void curob_dispose()
	{
		hipblasDestroy(handle);

		hipFree(o);
		hipFree(of);
		hipFree(m);
		hipFree(mf);
		hipFree(optima);
		hipFree(optimaf);

		hipFree(bufferA);
		hipFree(bufferAf);
		hipFree(bufferB);	
		hipFree(bufferBf);	
		hipFree(bufferC);
		hipFree(bufferCf);

		hipFree(omega);
		hipFree(omegaf);

		hipFree(devx);	
		hipFree(devxf);
		hipFree(devy);
		hipFree(devyf);

		CHECK_CUDA_ERROR();

		initialized = false;
	}

	int fids[] = {
		SPHERE, ELLIPSOID, ELLIPTIC, DISCUS, CIGAR, POWERS, SHARPV,
		STEP, WEIERSTRASS, GRIEWANK, RASTRIGIN_U, RASTRIGIN, SCHAFFERSF7, GRIE_ROSEN,
		ROSENBROCK, SCHWEFEL_U, SCHWEFEL, KATSUURA,  LUNACEK,  ACKLEY, HAPPYCAT, HGBAT, SCHAFFERSF6,
		HYBRID1, HYBRID2, HYBRID3, HYBRID4, HYBRID5, HYBRID6,
		COMPOSITION1, COMPOSITION2, COMPOSITION3, COMPOSITION4, COMPOSITION5, COMPOSITION6, COMPOSITION7, COMPOSITION8
	};

#ifdef __cplusplus
}
#endif


/*
** Single Precison Float
*/
#define float_t float
#define SUFFIX(name) name ## f
#define gemm hipblasSgemm
#define scal hipblasSscal
#define nrm2 hipblasSnrm2
#define axpy hipblasSaxpy
#include "cuROB_kernels.inc"


#undef float_t
#undef SUFFIX
#undef gemm
#undef scal
#undef nrm2
#undef axpy


/*
** Double Precision Float
*/
#define float_t double
#define SUFFIX(name) name
#define gemm hipblasDgemm
#define scal hipblasDscal
#define nrm2 hipblasDnrm2
#define axpy hipblasDaxpy
#include "cuROB_kernels.inc"




